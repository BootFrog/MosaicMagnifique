#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/hip_math_constants.h>

#include "cudaphotomosaicdata.h"
#include "reduction.cuh"

//Calculates the euclidean difference between main image and library images
__global__
void euclideanDifferenceKernel(uchar *im_1, uchar *im_2, size_t noLibIm, uchar *mask_im,
                               size_t size, size_t channels, size_t *target_area, double *variants)
{
    const size_t index = (blockIdx.x * blockDim.x + threadIdx.x) * channels;
    const size_t stride = blockDim.x * gridDim.x * channels;
    for (size_t i = index; i < size * size * channels * noLibIm; i += stride)
    {
        const size_t im_1_index = i % (size * size * channels);
        const size_t grayscaleIndex = im_1_index / channels;

        const size_t row = grayscaleIndex / size;
        if (row < target_area[0] || row >= target_area[1])
        {
            variants[i / channels] = 0;
            continue;
        }

        const size_t col = grayscaleIndex % size;
        if (col < target_area[2] || col >= target_area[3])
        {
            variants[i / channels] = 0;
            continue;
        }

        if (mask_im[grayscaleIndex] == 0)
            variants[i / channels] = 0;
        else
            variants[i / channels] = sqrt(pow((double) im_1[im_1_index] - im_2[i], 2.0) +
                                          pow((double) im_1[im_1_index + 1] - im_2[i + 1], 2.0) +
                                          pow((double) im_1[im_1_index + 2] - im_2[i + 2], 2.0));
    }
}

//Converts degrees to radians
__device__
constexpr double degToRadKernel(const double deg)
{
    return (deg * HIP_PI) / 180;
}

//Kernel that calculates the CIEDE2000 difference between main image and library images
__global__
void CIEDE2000DifferenceKernel(uchar *im_1, uchar *im_2, size_t noLibIm, uchar *mask_im,
                               size_t size, size_t channels, size_t *target_area, double *variants)
{
    const size_t index = (blockIdx.x * blockDim.x + threadIdx.x) * channels;
    const size_t stride = blockDim.x * gridDim.x * channels;
    for (size_t i = index; i < size * size * channels * noLibIm; i += stride)
    {
        const size_t im_1_index = i % (size * size * channels);
        const size_t grayscaleIndex = im_1_index / channels;

        const size_t row = grayscaleIndex / size;
        if (row < target_area[0] || row >= target_area[1])
        {
            variants[i / channels] = 0;
            continue;
        }

        const size_t col = grayscaleIndex % size;
        if (col < target_area[2] || col >= target_area[3])
        {
            variants[i / channels] = 0;
            continue;
        }

        if (mask_im[grayscaleIndex] == 0)
            variants[i / channels] = 0;
        else
        {
            const double k_L = 1.0, k_C = 1.0, k_H = 1.0;
            constexpr double deg360InRad = degToRadKernel(360.0);
            constexpr double deg180InRad = degToRadKernel(180.0);
            const double pow25To7 = 6103515625.0; //pow(25, 7)

            const double C1 = sqrt((double) (im_1[im_1_index + 1] * im_1[im_1_index + 1]) +
                    (im_1[im_1_index + 2] * im_1[im_1_index + 2]));
            const double C2 = sqrt((double) (im_2[i + 1] * im_2[i + 1]) +
                    (im_2[i + 2] * im_2[i + 2]));
            const double barC = (C1 + C2) / 2.0;

            const double G = 0.5 * (1 - sqrt(pow(barC, 7) / (pow(barC, 7) + pow25To7)));

            const double a1Prime = (1.0 + G) * im_1[im_1_index + 1];
            const double a2Prime = (1.0 + G) * im_2[i + 1];

            const double CPrime1 = sqrt((a1Prime * a1Prime) +
                                        (im_1[im_1_index + 2] * im_1[im_1_index + 2]));
            const double CPrime2 = sqrt((a2Prime * a2Prime) +(im_2[i + 2] * im_2[i + 2]));

            double hPrime1;
            if (im_1[im_1_index + 2] == 0 && a1Prime == 0.0)
                hPrime1 = 0.0;
            else
            {
                hPrime1 = atan2((double) im_1[im_1_index + 2], a1Prime);
                //This must be converted to a hue angle in degrees between 0 and 360 by
                //addition of 2 pi to negative hue angles.
                if (hPrime1 < 0)
                    hPrime1 += deg360InRad;
            }

            double hPrime2;
            if (im_2[i + 2] == 0 && a2Prime == 0.0)
                hPrime2 = 0.0;
            else
            {
                hPrime2 = atan2((double) im_2[i + 2], a2Prime);
                //This must be converted to a hue angle in degrees between 0 and 360 by
                //addition of 2pi to negative hue angles.
                if (hPrime2 < 0)
                    hPrime2 += deg360InRad;
            }

            const double deltaLPrime = im_2[i] - im_1[im_1_index];
            const double deltaCPrime = CPrime2 - CPrime1;

            double deltahPrime;
            const double CPrimeProduct = CPrime1 * CPrime2;
            if (CPrimeProduct == 0.0)
                deltahPrime = 0;
            else
            {
                //Avoid the fabs() call
                deltahPrime = hPrime2 - hPrime1;
                if (deltahPrime < -deg180InRad)
                    deltahPrime += deg360InRad;
                else if (deltahPrime > deg180InRad)
                    deltahPrime -= deg360InRad;
            }

            const double deltaHPrime = 2.0 * sqrt(CPrimeProduct) * sin(deltahPrime / 2.0);

            const double barLPrime = (im_1[im_1_index] + im_2[i]) / 2.0;
            const double barCPrime = (CPrime1 + CPrime2) / 2.0;

            double barhPrime;
            const double hPrimeSum = hPrime1 + hPrime2;
            if (CPrime1 * CPrime2 == 0.0)
                barhPrime = hPrimeSum;
            else
            {
                if (fabs(hPrime1 - hPrime2) <= deg180InRad)
                    barhPrime = hPrimeSum / 2.0;
                else
                {
                    if (hPrimeSum < deg360InRad)
                        barhPrime = (hPrimeSum + deg360InRad) / 2.0;
                    else
                        barhPrime = (hPrimeSum - deg360InRad) / 2.0;
                }
            }

            const double T = 1.0 - (0.17 * cos(barhPrime - degToRadKernel(30.0))) +
                    (0.24 * cos(2.0 * barhPrime)) +
                    (0.32 * cos((3.0 * barhPrime) + degToRadKernel(6.0))) -
                    (0.20 * cos((4.0 * barhPrime) - degToRadKernel(63.0)));

            const double deltaTheta = degToRadKernel(30.0) *
                    exp(-pow((barhPrime - degToRadKernel(275.0)) / degToRadKernel(25.0), 2.0));

            const double R_C = 2.0 * sqrt(pow(barCPrime, 7.0) /
                                          (pow(barCPrime, 7.0) + pow25To7));

            const double S_L = 1 + ((0.015 * pow(barLPrime - 50.0, 2.0)) /
                                    sqrt(20 + pow(barLPrime - 50.0, 2.0)));
            const double S_C = 1 + (0.045 * barCPrime);
            const double S_H = 1 + (0.015 * barCPrime * T);

            const double R_T = (-sin(2.0 * deltaTheta)) * R_C;


            variants[i / channels] = (double) sqrt(pow(deltaLPrime / (k_L * S_L), 2.0) +
                                                   pow(deltaCPrime / (k_C * S_C), 2.0) +
                                                   pow(deltaHPrime / (k_H * S_H), 2.0) +
                                                   (R_T * (deltaCPrime / (k_C * S_C)) *
                                                    (deltaHPrime / (k_H * S_H))));
        }
    }
}

//Calculates repeats in range
__global__
void calculateRepeats(bool *states, size_t *bestFit, size_t *repeats,
                      const int noXCell,
                      const int leftRange, const int rightRange,
                      const int upRange,
                      const size_t repeatAddition)
{
    for (int y = -upRange; y < 0; ++y)
    {
        for (int x = -leftRange; x <= rightRange; ++x)
        {
            if (states[y * noXCell + x])
                repeats[bestFit[y * noXCell + x]] += repeatAddition;
        }
    }
    for (int x = -leftRange; x < 0; ++x)
    {
        if (states[x])
            repeats[bestFit[x]] += repeatAddition;
    }
}

//Adds repeat values to variants
__global__
void addRepeatsKernel(double *variants, size_t *repeats, size_t noLibIm)
{
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < noLibIm; i += stride)
        variants[i] += repeats[i];
}

//Finds lowest value in variants
__global__
void findLowestKernel(double *lowestVariant, size_t *bestFit, double *variants, size_t noLibIm)
{
    for (size_t i = 0; i < noLibIm; ++i)
    {
        if (variants[i] < *lowestVariant)
        {
            *lowestVariant = variants[i];
            *bestFit = i;
        }
    }
}

size_t differenceGPU(CUDAPhotomosaicData &photomosaicData)
{
    size_t numBlocks;
    const int batchSize = static_cast<int>(photomosaicData.getBatchSize());
    int batchIndex;

    //Create streams
    const static size_t maxStreams = 8;
    const size_t noOfStreams = std::min(maxStreams, photomosaicData.getBatchSize());
    size_t curStream = 0;
    hipStream_t streams[maxStreams];
    for (size_t i = 0; i < noOfStreams; ++i)
        gpuErrchk(hipStreamCreate(&streams[i]));

    //Loop over all batches
    while ((batchIndex = photomosaicData.copyNextBatchToDevice()) != -1)
    {
        //Loop over all data in batch
        for (size_t i = 0; i < batchSize
             && batchIndex * batchSize + i < photomosaicData.noCellImages; ++i)
        {
            //Skip if cell invalid
            if (!photomosaicData.getCellState(i))
                continue;

            const int x = (batchIndex * batchSize + static_cast<int>(i))
                    % static_cast<int>(photomosaicData.noXCellImages);
            const int y = (batchIndex * batchSize + static_cast<int>(i))
                    / static_cast<int>(photomosaicData.noXCellImages);

            //Calculate differences
            numBlocks = (photomosaicData.pixelCount * photomosaicData.noLibraryImages
                         + photomosaicData.getBlockSize() - 1) / photomosaicData.getBlockSize();
            if (photomosaicData.euclidean)
                euclideanDifferenceKernel<<<static_cast<unsigned int>(numBlocks),
                        static_cast<unsigned int>(photomosaicData.getBlockSize()),
                        0, streams[curStream]>>>(
                                photomosaicData.getCellImage(i),
                                photomosaicData.getLibraryImages(), photomosaicData.noLibraryImages,
                                photomosaicData.getMaskImage(x, y),
                                photomosaicData.imageSize, photomosaicData.imageChannels,
                                photomosaicData.getTargetArea(i),
                                photomosaicData.getVariants(i));
            else
                CIEDE2000DifferenceKernel<<<static_cast<unsigned int>(numBlocks),
                        static_cast<unsigned int>(photomosaicData.getBlockSize()),
                        0, streams[curStream]>>>(
                                photomosaicData.getCellImage(i),
                                photomosaicData.getLibraryImages(), photomosaicData.noLibraryImages,
                                photomosaicData.getMaskImage(x, y),
                                photomosaicData.imageSize, photomosaicData.imageChannels,
                                photomosaicData.getTargetArea(i),
                                photomosaicData.getVariants(i));

            //Move to next stream
            ++curStream;
            if (curStream == noOfStreams)
                curStream = 0;
        }
        //Perform sum reduction on all image variants
        reduceAddData(photomosaicData, streams, noOfStreams);

        //Loop over all data in batch
        for (size_t i = 0; i < batchSize
             && batchIndex * batchSize + i < photomosaicData.noCellImages; ++i)
        {
            //Skip if cell invalid
            if (!photomosaicData.getCellState(i))
                continue;

            //Calculate repeats
            photomosaicData.clearRepeats();
            const int x = (batchIndex * batchSize + static_cast<int>(i))
                    % static_cast<int>(photomosaicData.noXCellImages);
            const int y = (batchIndex * batchSize + static_cast<int>(i))
                    / static_cast<int>(photomosaicData.noXCellImages);

            const int leftRange = std::min(static_cast<int>(photomosaicData.repeatRange), x);
            const int rightRange = std::min(static_cast<int>(photomosaicData.repeatRange),
                                            static_cast<int>(photomosaicData.noXCellImages)
                                            - x - 1);
            const int upRange = std::min(static_cast<int>(photomosaicData.repeatRange), y);
            calculateRepeats<<<1, 1>>>(photomosaicData.getCellStateGPU(i),
                                       photomosaicData.getBestFit(i), photomosaicData.getRepeats(),
                                       static_cast<int>(photomosaicData.noXCellImages),
                                       leftRange, rightRange, upRange,
                                       photomosaicData.repeatAddition);

            //Adds repeat values to differences
            numBlocks = (photomosaicData.noLibraryImages
                         + photomosaicData.getBlockSize() - 1) / photomosaicData.getBlockSize();
            addRepeatsKernel<<<static_cast<unsigned int>(numBlocks),
                    static_cast<unsigned int>(photomosaicData.getBlockSize())>>>(
                            photomosaicData.getVariants(i),
                            photomosaicData.getRepeats(),
                            photomosaicData.noLibraryImages);

            //Find lowest variant
            findLowestKernel<<<1, 1>>>(photomosaicData.getLowestVariant(i),
                                       photomosaicData.getBestFit(i),
                                       photomosaicData.getVariants(i),
                                       photomosaicData.noLibraryImages);
        }
    }
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    //Destroy streams
    for (size_t i = 0; i < noOfStreams; ++i)
        gpuErrchk(hipStreamDestroy(streams[i]));

    return 0;
}
